#include "SpMV.h"
#include "stdio.h"
/*
#include <hip/hip_runtime.h>  // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>      // hipsparseSpMV
#include <stdio.h>         // printf
#include <stdlib.h>        // EXIT_FAILURE

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}*/

#ifdef __cplusplus
#define EXTERNC extern "C"
#else
#define EXTERNC
#endif
/*
int64_t glob_m = 0;
hipsparseHandle_t     handle = 0;
hipsparseSpMatDescr_t matA;
hipsparseDnVecDescr_t vecX, vecY;
void*  dBuffer    = NULL;
*/
SpMVDoubleWarp* spmv;

EXTERNC void createLightSpMV(int64_t m, int64_t nnz, int64_t* drowptr, int64_t* dcolind, double* dvalues){ 
/*	glob_m = m;

    // CUSPARSE APIs
    size_t bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, m, m, nnz,
                                      drowptr, dcolind, dvalues,
                                      HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I,
                                      HIPSPARSE_INDEX_BASE_ONE, HIP_R_64F) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, m, NULL, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, m, NULL, HIP_R_64F) )
    // Create dense vector y
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUSPARSE( hipMalloc(&dBuffer, bufferSize) )
*/
	
	
	Options* opt = new Options;
	opt->_numRows = m;
	opt->_numCols = m;
	opt->_numValues = nnz;
	if(!opt->getGPUs())
		printf("Error while scanning for GPU\n");
	
	spmv = new SpMVDoubleWarp(opt);
	//hipDeviceSetCacheConfig (hipFuncCachePreferL1);
	spmv->loadData();
}

EXTERNC void performLightLsMV(
    double alpha,
    double* dval,
    int64_t* drowptr,
    int64_t* dcolind,
    double* dx,
    double beta,
    double* dy){
		
/*
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, m, dx, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, m, dy, HIP_R_64F) )
		
		
    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
*/
	
	spmv->_rowOffsets[0] = drowptr;
	spmv->_colIndexValues[0] = dcolind;
	spmv->_numericalValues[0] = dval;
	spmv->_vectorX[0] = dx;
	spmv->_vectorY[0] = dy;
	
	spmv->_alpha = alpha;
	spmv->_beta = beta;
	
	spmv->spmvKernel();
}

EXTERNC void destroyLightSpMV(){
	/*
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    
    CHECK_CUDA( hipFree(dBuffer) )
	*/
	
	
	spmv->_rowOffsets[0] = NULL;
	spmv->_colIndexValues[0] = NULL;
	spmv->_numericalValues[0] = NULL;
	delete spmv;
}

#undef EXTERNC
